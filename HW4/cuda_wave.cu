/**********************************************************************
 * DESCRIPTION:
 *   Serial Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

void check_param(void);
void printfinal (void);

int nsteps,                 	/* number of time steps */
    tpoints, 	     		/* total points along string */
    rcode;                  	/* generic return code */
float  values[MAXPOINTS+2]; 	/* values at time t */


/**********************************************************************
 *	Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
   char tchar[20];

   /* check number of points, number of iterations */
   while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
      printf("Enter number of points along vibrating string [%d-%d]: "
           ,MINPOINTS, MAXPOINTS);
      scanf("%s", tchar);
      tpoints = atoi(tchar);
      if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
         printf("Invalid. Please enter value between %d and %d\n", 
                 MINPOINTS, MAXPOINTS);
   }
   while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
      printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
      scanf("%s", tchar);
      nsteps = atoi(tchar);
      if ((nsteps < 1) || (nsteps > MAXSTEPS))
         printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
   }

   printf("Using points = %d, steps = %d\n", tpoints, nsteps);

}



/**********************************************************************
 *     Update all values along line a specified number of times
 *********************************************************************/
__global__ void kernel_update(float *val, int tpoints, int nsteps, int start_point)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x + 1 + start_point;
    __shared__ float x, fac, d_oldval, d_newval, d_tmpval;
   	__shared__ float dtime, c, dx, tau, sqtau;

    //--------------------------------------------------------------
    // init line (moved into update())
    //--------------------------------------------------------------
    fac = 2.0 * PI;
	x = (idx - 1.0) / (float)(tpoints - 1);
	d_tmpval = sin(fac * x);	
	d_oldval = d_tmpval;

    //--------------------------------------------------------------
    // math paramerters setup (moved into update())
    //--------------------------------------------------------------
   	dtime = 0.3;
   	c = 1.0;
   	dx = 1.0;
   	tau = (c * dtime / dx);
   	sqtau = tau * tau;

    if ((idx == 1) || (idx == tpoints)){
       	d_newval = 0.0;
    }
    /* Update values for each time step */
    for (int i = 1; i<= nsteps; i++) {
       	d_newval = (2.0 * d_tmpval) - d_oldval + (sqtau * (-2.0) * d_tmpval);
        d_oldval = d_tmpval;
        d_tmpval = d_newval;
    }
	if (start_point != 0)
    	val[idx - start_point - 1] = d_tmpval;
	else
    	val[idx] = d_tmpval;
}

/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal()
{
   int i;

   for (i = 1; i <= tpoints; i++) {
      printf("%6.4f ", values[i]);
      if (i%10 == 0)
         printf("\n");
   }
}

/**********************************************************************
 *	Main program
 *********************************************************************/
int main(int argc, char *argv[])
{   
	sscanf(argv[1],"%d",&tpoints);
	sscanf(argv[2],"%d",&nsteps);
	check_param();
	printf("Initializing points on the line...\n");
	//init_line();
	printf("Updating all points for all time steps...\n");

    //--------------------------------------------------------------
    // Launch Kernal
    //--------------------------------------------------------------
    float *d_val;
	float *d_remain;
    int tile_width = 1024;
	int remain = tpoints % tile_width;  
    dim3 dimBlock(tile_width);
    dim3 dimGrid(tpoints / tile_width);
    hipMalloc((void**)&d_val, (tpoints + 1) * sizeof(float));
    hipMalloc((void**)&d_remain, (remain + 1) * sizeof(float));
	// two streams so kernel function could overlap
	hipStream_t stream1, stream2;
	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);
	kernel_update<<<dimGrid, dimBlock, 0, stream1>>>(d_val, tpoints, nsteps, 0);
	// second call is for the remainder
	kernel_update<<<1, remain, 0, stream2>>>(d_remain, tpoints, nsteps, tpoints - remain - 1);
    //--------------------------------------------------------------
    // Data transfer back in async. manner 
    //--------------------------------------------------------------
    hipMemcpyAsync(values, d_val, (tpoints + 1)  * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpyAsync(values + tpoints - remain, d_remain, (remain + 1) * sizeof(float), hipMemcpyDeviceToHost);
	hipStreamDestroy(stream1);
	hipStreamDestroy(stream2);
	printf("Printing final results...\n");
	printfinal();
	printf("\nDone.\n\n");
    hipFree(d_val);
    hipFree(d_remain);
	return 0;
}
